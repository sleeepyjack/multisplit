#include <iostream>
#include <assert.h>
#include <omp.h>

#include "cuda_helpers.cuh"
#include "hash_functions.cuh"

template<class Index,
         Index K,
         class SplitFunc>
GLOBALQUALIFIER void multisplit(Index * input,
                                Index ** splits,
                                Index * split_counts,
                                Index len,
                                SplitFunc split_func)
{
    for(Index tid = blockIdx.x * blockDim.x + threadIdx.x;
        tid < len;
        tid += gridDim.x * blockDim.x)
    {
        const Index value    = input[tid];
        const Index my_split = split_func(value) % K;

        #pragma unroll K
        for(Index i = 0; i < K; ++i)
        {
            if(i == my_split)
            {
                const Index j = atomicAggInc(&split_counts[i]);
                splits[i][j]  = value;
            }
        }
    }
}

template<class Index,
         Index K,
         class SplitFunc>
GLOBALQUALIFIER void validate(Index ** split,
                              Index * split_count,
                              Index this_split,
                              SplitFunc split_func)
{
    const Index tid = blockIdx.x*blockDim.x+threadIdx.x;
    if(tid >= split_count[this_split]) return;

    const Index value = split[this_split][tid];
    const Index my_split = split_func(value) % K;

    assert(my_split == this_split); //or die!
}

int main()
{
    using index_t = unsigned int; //the input and index type to be used
    using split_t = mueller_hash_uint32_t; //the split function

    //PARAMETERS
    constexpr index_t len = (1UL<<28); //num input elements
    constexpr index_t k   = 4; //num splits

    std::cout << "PARAMS: input_length: " << len
              << "\t"   << "num_splits: " << k
                                          << std::endl;

    TIMERSTART(init)
    //init K split arrays
    index_t ** splits_h = new index_t*[k];
    for(index_t i = 0; i < k; ++i)
    {
        hipMalloc(&splits_h[i], sizeof(index_t)*len); CUERR
    }
    index_t ** splits_d; hipMalloc(&splits_d, sizeof(index_t*)*k); CUERR
    hipMemcpy(splits_d, splits_h, sizeof(index_t*)*k, H2D); CUERR

    //init split counters
    index_t * split_counts; hipMalloc(&split_counts, sizeof(index_t)*k); CUERR
    memset_kernel<<<SDIV(len, 1024), 1024>>>(split_counts, k, index_t(0)); CUERR

    //init input array
    index_t * input_h = new index_t[len];
    #pragma omp parallel for
    for(index_t i = 0; i < len; ++i)
    {
        input_h[i] = i;
    }
    index_t * input_d; hipMalloc(&input_d, sizeof(index_t)*len); CUERR
    hipMemcpy(input_d, input_h, sizeof(index_t)*len, H2D); CUERR
    TIMERSTOP(init)


    //execute multisplit
    TIMERSTART(multisplit)
    multisplit
    <index_t, k, split_t>
    <<<SDIV(len, 1024), 1024>>>
    (input_d, splits_d, split_counts, len, split_t()); CUERR
    hipDeviceSynchronize(); CUERR
    TIMERSTOP(multisplit) CUERR
    float input_gb  = sizeof(index_t)*len/1000000000.0f; //input size in GB
    float time_sec  = timemultisplit/1000.0f; //exec time in seconds
    float bandwidth = input_gb/time_sec;
    std::cout << "BANDWIDTH: " << bandwidth << " GB/s" << std::endl;

    //validate results
    TIMERSTART(validate)
    for (index_t i = 0; i < k; ++i)
    {
        validate
        <index_t, k, split_t>
        <<<SDIV(len, 1024), 1024>>>
        (splits_d, split_counts, i, split_t()); CUERR
    }
    hipDeviceSynchronize(); CUERR
    TIMERSTOP(validate)

    //free memory
    delete[] splits_h;
    for(index_t i = 0; i < k; ++i)
    {
        hipFree(splits_h[i]); CUERR
    }
    hipFree(splits_d); CUERR

    hipFree(split_counts); CUERR

    delete[] input_h;
    hipFree(input_d); CUERR
}
